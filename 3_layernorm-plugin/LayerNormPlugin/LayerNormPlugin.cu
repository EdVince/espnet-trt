#include "hip/hip_runtime.h"
#include "LayerNormPlugin.h"

using namespace nvinfer1;

PluginFieldCollection    LayerNormPluginCreator::fc_ {};
std::vector<PluginField> LayerNormPluginCreator::attr_;

template<typename T, int n>
__global__ void layerNormKernel(T *pInput, T *pOutput, float epsilon)
{
    const int tx = threadIdx.x, index = blockIdx.x * n + threadIdx.x;

    T _x = pInput[index];

    __shared__ T mean_shared, var_shared;

    typedef hipcub::BlockReduce<T, n>               BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp;
    T &                                          ref0 = _x;
    T                                            sum  = BlockReduce(temp).Sum(ref0);

    if (tx == 0)
        mean_shared = sum / (T)n;
    __syncthreads();

    T  moment = _x - mean_shared, moment2 = moment * moment;
    T &ref1 = moment2;
    T  var  = BlockReduce(temp).Sum(ref1);

    if (tx == 0)
        var_shared = var / (T)n;
    __syncthreads();

    pOutput[index] = moment * (T)rsqrtf(var_shared + (T)epsilon);
}

int32_t LayerNormPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept
{
    WHERE_AM_I();
    int nBlock = inputDesc[0].dims.d[0] * inputDesc[0].dims.d[1];
    int nValuePerBlock = inputDesc[0].dims.d[2];

    if (inputDesc[0].type == DataType::kFLOAT)
    {
        switch (nValuePerBlock)
        {
        case 192:
            (layerNormKernel<float, 192>)<<<nBlock, 192, 0, stream>>>((float *)inputs[0], (float *)outputs[0], epsilon_);
            break;
        default: // shoulf NOT be here
            printf("[LayerNormPlugin::enqueue] nValuePerBlock = %d is not supported\n", nValuePerBlock);
            break;
        }
    }
    else
    {
        switch (nValuePerBlock)
        {
        case 192:
            (layerNormKernel<half, 192>)<<<nBlock, 192, 0, stream>>>((half *)inputs[0], (half *)outputs[0], epsilon_);
            break;
        default: // shoulf NOT be here
            printf("[LayerNormPlugin::enqueue] nValuePerBlock = %d is not supported\n", nValuePerBlock);
            break;
        }
    }
    return 0;
}

REGISTER_TENSORRT_PLUGIN(LayerNormPluginCreator);