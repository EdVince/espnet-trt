#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "CSMHAPlugin.h"

#define ALIGNSIZE  1024
#define ALIGNED(x) (((x) + ALIGNSIZE - 1) / ALIGNSIZE * ALIGNSIZE)

namespace nvinfer1
{
// class CSMHAPlugin
CSMHAPlugin::CSMHAPlugin(const std::string &name, 
                            nvinfer1::Weights qweight, nvinfer1::Weights qbias, 
                            nvinfer1::Weights kweight, nvinfer1::Weights kbias, 
                            nvinfer1::Weights vweight, nvinfer1::Weights vbias, 
                            nvinfer1::Weights oweight, nvinfer1::Weights obias, 
                            nvinfer1::Weights qkbias, nvinfer1::Weights qcbias, 
                            int k, int n, bool needDeepCopy = false):
    name_(name), bOwnWeight_(needDeepCopy), nK_(k), nN_(n)
{
    WHERE_AM_I()
    
    {
        // Load Q weight and bias
        {
            assert(qweight.type == DataType::kFLOAT);
            assert(qweight.values != nullptr);
            assert(qweight.count == k * n);

            qweight_.type  = DataType::kFLOAT;
            qweight_.count = qweight.count;
            if (needDeepCopy)
            {
                size_t size    = sizeof(float) * qweight.count;
                qweight_.values = malloc(size);
                memcpy(reinterpret_cast<char *>(const_cast<void *>(qweight_.values)), qweight.values, size);
            }
            else
            {
                qweight_.values = qweight.values;
            }
        }
        {
            assert(qbias.type == DataType::kFLOAT);
            assert(qbias.values != nullptr);
            assert(qbias.count == n);

            qbias_.type  = DataType::kFLOAT;
            qbias_.count = qbias.count;
            if (needDeepCopy)
            {
                size_t size    = sizeof(float) * qbias.count;
                qbias_.values = malloc(size);
                memcpy(reinterpret_cast<char *>(const_cast<void *>(qbias_.values)), qbias.values, size);
            }
            else
            {
                qbias_.values = qbias.values;
            }
        }
    }

    {
        // Load K weight and bias
        {
            assert(kweight.type == DataType::kFLOAT);
            assert(kweight.values != nullptr);
            assert(kweight.count == k * n);

            kweight_.type  = DataType::kFLOAT;
            kweight_.count = kweight.count;
            if (needDeepCopy)
            {
                size_t size    = sizeof(float) * kweight.count;
                kweight_.values = malloc(size);
                memcpy(reinterpret_cast<char *>(const_cast<void *>(kweight_.values)), kweight.values, size);
            }
            else
            {
                kweight_.values = kweight.values;
            }
        }
        {
            assert(kbias.type == DataType::kFLOAT);
            assert(kbias.values != nullptr);
            assert(kbias.count == n);

            kbias_.type  = DataType::kFLOAT;
            kbias_.count = kbias.count;
            if (needDeepCopy)
            {
                size_t size    = sizeof(float) * kbias.count;
                kbias_.values = malloc(size);
                memcpy(reinterpret_cast<char *>(const_cast<void *>(kbias_.values)), kbias.values, size);
            }
            else
            {
                kbias_.values = kbias.values;
            }
        }
    }

    {
        // Load V weight and bias
        {
            assert(vweight.type == DataType::kFLOAT);
            assert(vweight.values != nullptr);
            assert(vweight.count == k * n);

            vweight_.type  = DataType::kFLOAT;
            vweight_.count = vweight.count;
            if (needDeepCopy)
            {
                size_t size    = sizeof(float) * vweight.count;
                vweight_.values = malloc(size);
                memcpy(reinterpret_cast<char *>(const_cast<void *>(vweight_.values)), vweight.values, size);
            }
            else
            {
                vweight_.values = vweight.values;
            }
        }
        {
            assert(vbias.type == DataType::kFLOAT);
            assert(vbias.values != nullptr);
            assert(vbias.count == n);

            vbias_.type  = DataType::kFLOAT;
            vbias_.count = vbias.count;
            if (needDeepCopy)
            {
                size_t size    = sizeof(float) * vbias.count;
                vbias_.values = malloc(size);
                memcpy(reinterpret_cast<char *>(const_cast<void *>(vbias_.values)), vbias.values, size);
            }
            else
            {
                vbias_.values = vbias.values;
            }
        }
    }

    {
        // Load V weight and bias
        {
            assert(oweight.type == DataType::kFLOAT);
            assert(oweight.values != nullptr);
            assert(oweight.count == k * n);

            oweight_.type  = DataType::kFLOAT;
            oweight_.count = oweight.count;
            if (needDeepCopy)
            {
                size_t size    = sizeof(float) * oweight.count;
                oweight_.values = malloc(size);
                memcpy(reinterpret_cast<char *>(const_cast<void *>(oweight_.values)), oweight.values, size);
            }
            else
            {
                oweight_.values = oweight.values;
            }
        }
        {
            assert(obias.type == DataType::kFLOAT);
            assert(obias.values != nullptr);
            assert(obias.count == n);

            obias_.type  = DataType::kFLOAT;
            obias_.count = obias.count;
            if (needDeepCopy)
            {
                size_t size    = sizeof(float) * obias.count;
                obias_.values = malloc(size);
                memcpy(reinterpret_cast<char *>(const_cast<void *>(obias_.values)), obias.values, size);
            }
            else
            {
                obias_.values = obias.values;
            }
        }
    }

    {
        // Load qk and qc bias
        {
            assert(qkbias.type == DataType::kFLOAT);
            assert(qkbias.values != nullptr);
            assert(qkbias.count == n);

            qkbias_.type  = DataType::kFLOAT;
            qkbias_.count = qkbias.count;
            if (needDeepCopy)
            {
                size_t size    = sizeof(float) * qkbias.count;
                qkbias_.values = malloc(size);
                memcpy(reinterpret_cast<char *>(const_cast<void *>(qkbias_.values)), qkbias.values, size);
            }
            else
            {
                qkbias_.values = qkbias.values;
            }
        }
        {
            assert(qcbias.type == DataType::kFLOAT);
            assert(qcbias.values != nullptr);
            assert(qcbias.count == n);

            qcbias_.type  = DataType::kFLOAT;
            qcbias_.count = qcbias.count;
            if (needDeepCopy)
            {
                size_t size    = sizeof(float) * qcbias.count;
                qcbias_.values = malloc(size);
                memcpy(reinterpret_cast<char *>(const_cast<void *>(qcbias_.values)), qcbias.values, size);
            }
            else
            {
                qcbias_.values = qcbias.values;
            }
        }
    }

    CHECK(hipblasCreate(&handle_));
}

CSMHAPlugin::CSMHAPlugin(const std::string &name, const void *buffer, size_t length):
    name_(name), bOwnWeight_(true)
{
    WHERE_AM_I()
    const char *data   = reinterpret_cast<const char *>(buffer);
    size_t      offset = 0;
    memcpy(&nK_, data + offset, sizeof(nK_));
    offset += sizeof(nK_);
    memcpy(&nN_, data + offset, sizeof(nN_));
    offset += sizeof(nN_);


    size_t size;

    {
        qweight_.type   = DataType::kFLOAT;
        qweight_.count  = nK_ * nN_;
        size            = sizeof(float) * nK_ * nN_;
        qweight_.values = malloc(size);
        memcpy(reinterpret_cast<char *>(const_cast<void *>(qweight_.values)), data + offset, size);
        offset += size;

        qbias_.type   = DataType::kFLOAT;
        qbias_.count  = nN_;
        size          = sizeof(float) * nN_;
        qbias_.values = malloc(size);
        memcpy(reinterpret_cast<char *>(const_cast<void *>(qbias_.values)), data + offset, size);
        offset += size;
    }

    {
        kweight_.type   = DataType::kFLOAT;
        kweight_.count  = nK_ * nN_;
        size            = sizeof(float) * nK_ * nN_;
        kweight_.values = malloc(size);
        memcpy(reinterpret_cast<char *>(const_cast<void *>(kweight_.values)), data + offset, size);
        offset += size;

        kbias_.type   = DataType::kFLOAT;
        kbias_.count  = nN_;
        size         = sizeof(float) * nN_;
        kbias_.values = malloc(size);
        memcpy(reinterpret_cast<char *>(const_cast<void *>(kbias_.values)), data + offset, size);
        offset += size;
    }

    {
        vweight_.type   = DataType::kFLOAT;
        vweight_.count  = nK_ * nN_;
        size            = sizeof(float) * nK_ * nN_;
        vweight_.values = malloc(size);
        memcpy(reinterpret_cast<char *>(const_cast<void *>(vweight_.values)), data + offset, size);
        offset += size;

        vbias_.type   = DataType::kFLOAT;
        vbias_.count  = nN_;
        size         = sizeof(float) * nN_;
        vbias_.values = malloc(size);
        memcpy(reinterpret_cast<char *>(const_cast<void *>(vbias_.values)), data + offset, size);
        offset += size;
    }

    {
        oweight_.type   = DataType::kFLOAT;
        oweight_.count  = nK_ * nN_;
        size            = sizeof(float) * nK_ * nN_;
        oweight_.values = malloc(size);
        memcpy(reinterpret_cast<char *>(const_cast<void *>(oweight_.values)), data + offset, size);
        offset += size;

        obias_.type   = DataType::kFLOAT;
        obias_.count  = nN_;
        size         = sizeof(float) * nN_;
        obias_.values = malloc(size);
        memcpy(reinterpret_cast<char *>(const_cast<void *>(obias_.values)), data + offset, size);
        offset += size;
    }

    {
        qkbias_.type   = DataType::kFLOAT;
        qkbias_.count  = nN_;
        size         = sizeof(float) * nN_;
        qkbias_.values = malloc(size);
        memcpy(reinterpret_cast<char *>(const_cast<void *>(qkbias_.values)), data + offset, size);
        offset += size;

        qcbias_.type   = DataType::kFLOAT;
        qcbias_.count  = nN_;
        size         = sizeof(float) * nN_;
        qcbias_.values = malloc(size);
        memcpy(reinterpret_cast<char *>(const_cast<void *>(qcbias_.values)), data + offset, size);
        offset += size;
    }

    CHECK(hipblasCreate(&handle_));
}

CSMHAPlugin::~CSMHAPlugin()
{
    WHERE_AM_I();
}

IPluginV2DynamicExt *CSMHAPlugin::clone() const noexcept
{
    WHERE_AM_I()
    CSMHAPlugin *p = new CSMHAPlugin(name_, 
                                        qweight_, qbias_, 
                                        kweight_, kbias_, 
                                        vweight_, vbias_, 
                                        oweight_, obias_, 
                                        qkbias_, qcbias_, 
                                        nK_, nN_, true);
    p->setPluginNamespace(namespace_.c_str());
    p->pGPUQWeight_ = this->pGPUQWeight_;
    p->pGPUQBias_ = this->pGPUQBias_;
    p->pGPUKWeight_ = this->pGPUKWeight_;
    p->pGPUKBias_ = this->pGPUKBias_;
    p->pGPUVWeight_ = this->pGPUVWeight_;
    p->pGPUVBias_ = this->pGPUVBias_;
    p->pGPUOWeight_ = this->pGPUOWeight_;
    p->pGPUOBias_ = this->pGPUOBias_;
    p->pGPUQKBias_ = this->pGPUQKBias_;
    p->pGPUQCBias_ = this->pGPUQCBias_;
    return p;
}

int32_t CSMHAPlugin::getNbOutputs() const noexcept
{
    WHERE_AM_I()
    return 1;
}

DataType CSMHAPlugin::getOutputDataType(int32_t index, DataType const *inputTypes, int32_t nbInputs) const noexcept
{
    WHERE_AM_I()
    return inputTypes[0];
}

DimsExprs CSMHAPlugin::getOutputDimensions(int32_t outputIndex, const DimsExprs *inputs, int32_t nbInputs, IExprBuilder &exprBuilder) noexcept
{
    WHERE_AM_I()
    // DimsExprs ret {inputs[0]};
    // ret.d[inputs[0].nbDims - 1] = exprBuilder.constant(nN_);
    // return ret;
    return inputs[0];
}

bool CSMHAPlugin::supportsFormatCombination(int32_t pos, const PluginTensorDesc *inOut, int32_t nbInputs, int32_t nbOutputs) noexcept
{
    WHERE_AM_I()
    switch (pos)
    {
        case 0: return inOut[0].type == DataType::kFLOAT && inOut[0].format == TensorFormat::kLINEAR;
        case 1: return inOut[1].type == DataType::kFLOAT && inOut[1].format == TensorFormat::kLINEAR;
        case 2: return inOut[2].type == DataType::kFLOAT && inOut[2].format == TensorFormat::kLINEAR;
        case 3: return inOut[3].type == DataType::kFLOAT && inOut[3].format == TensorFormat::kLINEAR;
        default: return false;
    }
    return false;
}

void CSMHAPlugin::configurePlugin(const DynamicPluginTensorDesc *in, int32_t nbInputs, const DynamicPluginTensorDesc *out, int32_t nbOutputs) noexcept
{
    WHERE_AM_I();
}

size_t CSMHAPlugin::getWorkspaceSize(const PluginTensorDesc *inputs, int32_t nbInputs, const PluginTensorDesc *outputs, int32_t nbOutputs) const noexcept
{
    WHERE_AM_I()

    return ALIGNED(
        10 * sizeof(float) * 2 * 192 * 192
    );

    return 0;
}


template<typename T, int BX, int TX>
__global__ void addBiasByCopyKernel(T *inOut, T *bias)
{
    const int spaceIndex = blockIdx.y * BX * TX + blockIdx.x * TX + threadIdx.x * 4;
    const int dataIndex = threadIdx.x * 4;

    reinterpret_cast<float4*>(inOut+spaceIndex)[0] = reinterpret_cast<float4*>(bias+dataIndex)[0];
}

template<typename T, int THREAD>
__global__ void addBiasKernel(T *pInput, T *pOutput, T *bias)
{
    const int tx = threadIdx.x, index = blockIdx.x * THREAD + threadIdx.x;
    T _x = pInput[index], _b = bias[tx];
    pOutput[index] = _x + _b;
}

template<typename T, int TY, int TX>
__global__ void faster_transposeQKernel(T *pInput, T *pOutput)
{
    const int by = blockIdx.y, bx = blockIdx.x;
    const int ty = threadIdx.y, tx = threadIdx.x;
    const int BX = gridDim.x;

    const int inIndex = by*(BX*TY*TX)+bx*(TY*TX)+ty*TX+tx*4;
    const int ouIndex = by*(TY*BX*TX)+ty*(BX*TX)+bx*TX+tx*4;

    reinterpret_cast<float4*>(pOutput+ouIndex)[0] = reinterpret_cast<float4*>(pInput+inIndex)[0];
}

template<typename T, int TY, int TX>
__global__ void faster_transposeIQKernel(T *pInput, T *pOutput)
{
    const int by = blockIdx.y, bx = blockIdx.x;
    const int ty = threadIdx.y, tx = threadIdx.x;
    const int BX = gridDim.x;

    const int inIndex = by*(TY*BX*TX)+ty*(BX*TX)+bx*TX+tx*4;
    const int ouIndex = by*(BX*TY*TX)+bx*(TY*TX)+ty*TX+tx*4;

    reinterpret_cast<float4*>(pOutput+ouIndex)[0] = reinterpret_cast<float4*>(pInput+inIndex)[0];
}

// (B, X, Y) -> (B, Y, X)
// https://github.com/Oneflow-Inc/oneflow/blob/f0e9d38b2ba4ac535fd6de5dbeca4e3d2051de23/oneflow/core/ep/cuda/primitive/permute.cu#L57
template<typename T, size_t num_dims, size_t tile_size>
__global__ void faster_transposeKKernel(void* src_ptr, void* dst_ptr, int rows, int cols, int num_tile_rows, int num_tile_cols, int32_t block_nums)
{
    const int src_rows = rows;
    const int src_cols = cols;
    const int dst_rows = cols;
    const int dst_cols = rows;

    __shared__ T tile[tile_size][tile_size + 1];

    const T* src = reinterpret_cast<const T*>(src_ptr);
    T* dst = reinterpret_cast<T*>(dst_ptr);

    int batch_num_tile = num_tile_rows * num_tile_cols;
    for (int i = blockIdx.x, step = gridDim.x; i < block_nums; i += step) 
    {
        const int batch_index = i / batch_num_tile;
        const int tile_index = i - batch_index * batch_num_tile;

        const int tile_row_index = tile_index / num_tile_cols;
        const int tile_col_index = tile_index - tile_row_index * num_tile_cols;

        const int offset = batch_index * src_rows * src_cols;
        {
            int col_in_tile = threadIdx.x;
            int col_in_matrix = tile_col_index * tile_size + threadIdx.x;
#pragma unroll
            for (int row_in_tile = threadIdx.y; row_in_tile < tile_size; row_in_tile += 8) 
            {
                int row_in_matrix = row_in_tile + tile_row_index * tile_size;
                if (col_in_matrix < src_cols && row_in_matrix < src_rows) 
                {
                    tile[row_in_tile][col_in_tile] = src[offset + row_in_matrix * src_cols + col_in_matrix];
                }
            }
        }
        __syncthreads();
        {
            int col_in_tile = threadIdx.x;
            int col_in_matrix = tile_row_index * tile_size + threadIdx.x;
#pragma unroll
            for (int row_in_tile = threadIdx.y; row_in_tile < tile_size; row_in_tile += 8) 
            {
                int row_in_matrix = row_in_tile + tile_col_index * tile_size;
                if (col_in_matrix < dst_cols && row_in_matrix < dst_rows) 
                {
                    dst[offset + row_in_matrix * dst_cols + col_in_matrix] = tile[col_in_tile][row_in_tile];
                }
            }
        }
        __syncthreads();
    }
}

template<typename T>
__global__ void concat1Kernel(T *pInput, T *pOutput, int m)
{
    const int bx = blockIdx.x, tx = threadIdx.x;
    const int ouIndex = bx * 2 * m + tx;
    
    if (tx == 0)
        pOutput[ouIndex] = 0;
    else
        pOutput[ouIndex] = pInput[ouIndex-bx-1];
}

template<typename T>
__global__ void slice1Kernel(T *pInput, T *pOutput, int m)
{
    const int by = blockIdx.y, bx = blockIdx.x, tx = threadIdx.x;

    const int ouIndex = by*(2*m*m-m) + bx*m   + tx;
    const int inIndex = by*(2*m*m)   + bx*m+m + tx;

    pOutput[ouIndex] = pInput[inIndex];
}

template<typename T>
__global__ void slice2Kernel(T *pInput, T *pOutput, int m)
{
    const int bx = blockIdx.x, tx = threadIdx.x;

    const int ouIndex = bx*m + tx;
    const int inIndex = bx*(2*m-1) + tx;

    pOutput[ouIndex] = pInput[inIndex];
}

template<typename T>
__global__ void addScaleKernel(T *p1Input, T *p2Input, int m)
{
    const int bx = blockIdx.x, ty = threadIdx.y, tx = threadIdx.x;
    const int inOuIndex = bx*m*m + ty*m + tx;

    p2Input[inOuIndex] = (p1Input[inOuIndex] + p2Input[inOuIndex]) / 9.797959327697754f;
}

template<typename T>
__global__ void scaleMaskKernel(T *p1Input, T *p2Input, T *pMask, T *pOutput, int m)
{
    const int bx = blockIdx.x, tx = threadIdx.x;
    const int inOuIndex = bx*m + tx;

    if(pMask[tx] == 0)
        pOutput[inOuIndex] = -3.4028234663852886e+38f;
    else
        pOutput[inOuIndex] = (p1Input[inOuIndex] + p2Input[inOuIndex]) / 9.797959327697754f;
}

template<typename T>
__global__ void rowwiseMaxKernel(const T *pInput, T *pOutput, int m)
{
    const int bx = blockIdx.x, tx = threadIdx.x;
    const int index = bx*m + tx;

    T _x = -3.4028234663852886e+38f;
    if(tx<m)
        _x = pInput[index];

    typedef hipcub::BlockReduce<T, 144>              BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp;
    T                                            max  = BlockReduce(temp).Reduce(_x, hipcub::Max());

    if(tx == 0)
    {
        pOutput[bx] = max;
    }
}

template<typename T>
__global__ void expKernel(const T *pInput, const T *pMax, T *pOutput, int m)
{
    const int bx = blockIdx.x, tx = threadIdx.x;
    const int index = bx*m + tx;

    T _x = pInput[index];
    T _m = pMax[bx];

    pOutput[index] = expf(_x - _m);
}

template<typename T>
__global__ void rowwiseSumKernel(const T *pInput, T *pOutput, int m)
{
    const int bx = blockIdx.x, tx = threadIdx.x;
    const int index = bx*m + tx;

    T _x = 0.0f;
    if(tx<m)
        _x = pInput[index];

    typedef hipcub::BlockReduce<T, 144>              BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp;
    T                                            max  = BlockReduce(temp).Reduce(_x, hipcub::Sum());

    if(tx == 0)
    {
        pOutput[bx] = max;
    }
}

template<typename T>
__global__ void divKernel(const T *pInput, const T *pSum, T *pOutput, int m)
{
    const int tx = threadIdx.x, bx = blockIdx.x;
    const int index = bx*m + tx;

    T _x = pInput[index];
    T _s = pSum[bx];

    pOutput[index] = _x / _s;
}

int32_t CSMHAPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept
{
    WHERE_AM_I()

    if (nK_ == 192 && nN_ == 192 && 
        inputDesc[0].dims.nbDims == 3 && inputDesc[0].dims.d[0] == 1 && inputDesc[0].dims.d[2] == 192 &&
        inputDesc[1].dims.nbDims == 4 && inputDesc[1].dims.d[0] == 1 && inputDesc[1].dims.d[1] == 2 && inputDesc[1].dims.d[2] == 96 &&
        inputDesc[2].dims.nbDims == 3 && inputDesc[2].dims.d[0] == 1 && inputDesc[2].dims.d[1] == 1
    )
    {

        const int   m = inputDesc[0].dims.d[1];
        const int   mm = inputDesc[1].dims.d[3];
        const float alpha1 = 1.0f, beta0 = 0.0f, beta1 = 1.0f;
        const int   G = sizeof(float) * 2 * 192 * 192;

        // Q
        (addBiasByCopyKernel<float, 1, 192>)<<<dim3(1,m,1), dim3(48,1,1), 0, stream>>>((float *)(workspace+0*G), pGPUQBias_);
        CHECK(hipblasSgemm(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, nN_, m, nK_, &alpha1, pGPUQWeight_, nN_, (const float *)inputs[0], nK_, &beta1, (float *)(workspace+0*G), nN_));
        // 0

        // K
        (addBiasByCopyKernel<float, 1, 192>)<<<dim3(1,m,1), dim3(48,1,1), 0, stream>>>((float *)(workspace+1*G), pGPUKBias_);
        CHECK(hipblasSgemm(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, nN_, m, nK_, &alpha1, pGPUKWeight_, nN_, (const float *)inputs[0], nK_, &beta1, (float *)(workspace+1*G), nN_));
        // 0,1

        // V
        (addBiasByCopyKernel<float, 1, 192>)<<<dim3(1,m,1), dim3(48,1,1), 0, stream>>>((float *)(workspace+2*G), pGPUVBias_);
        CHECK(hipblasSgemm(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, nN_, m, nK_, &alpha1, pGPUVWeight_, nN_, (const float *)inputs[0], nK_, &beta1, (float *)(workspace+2*G), nN_));
        // 0,1,2

        // QK QC
        (addBiasKernel<float, 192>)<<<m, 192, 0, stream>>>((float *)(workspace+0*G), (float *)(workspace+3*G), pGPUQKBias_);
        (addBiasKernel<float, 192>)<<<m, 192, 0, stream>>>((float *)(workspace+0*G), (float *)(workspace+4*G), pGPUQCBias_);
        (faster_transposeQKernel<float, 2, 96>)<<<dim3(m,1,1), dim3(24,2,1), 0, stream>>>((float *)(workspace+3*G), (float *)(workspace+5*G));
        (faster_transposeQKernel<float, 2, 96>)<<<dim3(m,1,1), dim3(24,2,1), 0, stream>>>((float *)(workspace+4*G), (float *)(workspace+6*G));
        int tile_size = 16;
        int rows = m;
        int cols = 192;
        int num_tile_rows = (rows + tile_size - 1) / tile_size;
        int num_tile_cols = (cols + tile_size - 1) / tile_size;
        int32_t block_nums = 1 * num_tile_rows * num_tile_cols;
        (faster_transposeKKernel<float, 3, 16>)<<<block_nums, dim3(16, 8), 0, stream>>>((float *)(workspace+1*G), (float *)(workspace+7*G), rows, cols, num_tile_rows, num_tile_cols, block_nums);
        hipblasSgemmStridedBatched(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, m, m, 96, &alpha1, (float *)(workspace+7*G), m, 96*m, (float *)(workspace+5*G), 96, 96*m, &beta0, (float *)(workspace+8*G), m, m*m, 2);
        hipblasSgemmStridedBatched(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, 2*m-1, m, 96, &alpha1, (float *)(inputs[1]), 2*m-1, 96*(2*m-1), (float *)(workspace+6*G), 96, 96*m, &beta0, (float *)(workspace+9*G), 2*m-1, m*(2*m-1), 2);

        // Slice
        (concat1Kernel<float>)<<<2*m,2*m,0,stream>>>((float *)(workspace+9*G),(float *)(workspace+0*G),m);
        (slice1Kernel<float>)<<<dim3(2*m-1,2,1), m, 0, stream>>>((float *)(workspace+0*G),(float *)(workspace+1*G),m);
        (slice2Kernel<float>)<<<2*m, m, 0, stream>>>((float *)(workspace+1*G),(float *)(workspace+3*G),m);

        // AddMaskSoftmax
        (scaleMaskKernel<float>)<<<2*m, m, 0, stream>>>((float *)(workspace+8*G),(float *)(workspace+3*G),(float *)(inputs[2]),(float *)(workspace+3*G),m);
        (rowwiseMaxKernel<float>)<<<2*m, 144, 0, stream>>>((float *)(workspace+3*G),(float *)(workspace+1*G),m);
        (expKernel<float>)<<<2*m, m, 0, stream>>>((float *)(workspace+3*G),(float *)(workspace+1*G),(float *)(workspace+3*G),m);
        (rowwiseSumKernel<float>)<<<2*m, 144, 0, stream>>>((float *)(workspace+3*G),(float *)(workspace+1*G),m);
        (divKernel<float>)<<<2*m, m, 0, stream>>>((float *)(workspace+3*G),(float *)(workspace+1*G),(float *)(workspace+3*G),m);

        // // QKV
        (faster_transposeQKernel<float, 2, 96>)<<<dim3(m,1,1), dim3(24,2,1), 0, stream>>>((float *)(workspace+2*G), (float *)(workspace+4*G));
        hipblasSgemmStridedBatched(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, 96, m, m, &alpha1, (float *)(workspace+4*G), 96, 96*m, (float *)(workspace+3*G), m, m*m, &beta0, (float *)(workspace+5*G), 96, m*96, 2);
        (faster_transposeIQKernel<float, 2, 96>)<<<dim3(m,1,1), dim3(24,2,1), 0, stream>>>((float *)(workspace+5*G), (float *)(workspace+6*G));

        // // O
        (addBiasByCopyKernel<float, 1, 192>)<<<dim3(1,m,1), dim3(48,1,1), 0, stream>>>((float *)(outputs[0]), pGPUOBias_);
        CHECK(hipblasSgemm(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, nN_, m, nK_, &alpha1, pGPUOWeight_, nN_, (float *)(workspace+6*G), nK_, &beta1, (float *)(outputs[0]), nN_));
    }
    else
    {
        printf("[Kernel] [CSMHA] [CSMHAPlugin] [enqueue] only support input with (1,1,192) (1,2,96,1) (1,1,1), weight with (192,192) and bias with (192)\n");
        printf("[Kernel] [CSMHA] [CSMHAPlugin] [enqueue] but %d(%d,%d,%d) %d(%d,%d,%d,%d) %d(%d,%d,%d)\n",
            inputDesc[0].dims.nbDims,inputDesc[0].dims.d[0],inputDesc[0].dims.d[1],inputDesc[0].dims.d[2],
            inputDesc[1].dims.nbDims,inputDesc[1].dims.d[0],inputDesc[1].dims.d[1],inputDesc[1].dims.d[2],inputDesc[1].dims.d[3],
            inputDesc[2].dims.nbDims,inputDesc[2].dims.d[0],inputDesc[2].dims.d[1],inputDesc[2].dims.d[2]
        );
    }


    return 0;
}

int32_t CSMHAPlugin::initialize() noexcept
{
    WHERE_AM_I()

    size_t size;

    {
        size = sizeof(float) * qweight_.count;
        CHECK(hipMalloc((void **)&pGPUQWeight_, size));
        CHECK(hipMemcpy(pGPUQWeight_, qweight_.values, size, hipMemcpyHostToDevice));

        size = sizeof(float) * qbias_.count;
        CHECK(hipMalloc((void **)&pGPUQBias_, size));
        CHECK(hipMemcpy(pGPUQBias_, qbias_.values, size, hipMemcpyHostToDevice));
    }

    {
        size = sizeof(float) * kweight_.count;
        CHECK(hipMalloc((void **)&pGPUKWeight_, size));
        CHECK(hipMemcpy(pGPUKWeight_, kweight_.values, size, hipMemcpyHostToDevice));

        size = sizeof(float) * kbias_.count;
        CHECK(hipMalloc((void **)&pGPUKBias_, size));
        CHECK(hipMemcpy(pGPUKBias_, kbias_.values, size, hipMemcpyHostToDevice));
    }

    {
        size = sizeof(float) * vweight_.count;
        CHECK(hipMalloc((void **)&pGPUVWeight_, size));
        CHECK(hipMemcpy(pGPUVWeight_, vweight_.values, size, hipMemcpyHostToDevice));

        size = sizeof(float) * vbias_.count;
        CHECK(hipMalloc((void **)&pGPUVBias_, size));
        CHECK(hipMemcpy(pGPUVBias_, vbias_.values, size, hipMemcpyHostToDevice));
    }

    {
        size = sizeof(float) * oweight_.count;
        CHECK(hipMalloc((void **)&pGPUOWeight_, size));
        CHECK(hipMemcpy(pGPUOWeight_, oweight_.values, size, hipMemcpyHostToDevice));

        size = sizeof(float) * obias_.count;
        CHECK(hipMalloc((void **)&pGPUOBias_, size));
        CHECK(hipMemcpy(pGPUOBias_, obias_.values, size, hipMemcpyHostToDevice));
    }

    {
        size = sizeof(float) * qkbias_.count;
        CHECK(hipMalloc((void **)&pGPUQKBias_, size));
        CHECK(hipMemcpy(pGPUQKBias_, qkbias_.values, size, hipMemcpyHostToDevice));

        size = sizeof(float) * qcbias_.count;
        CHECK(hipMalloc((void **)&pGPUQCBias_, size));
        CHECK(hipMemcpy(pGPUQCBias_, qcbias_.values, size, hipMemcpyHostToDevice));
    }

    return 0;
}

void CSMHAPlugin::terminate() noexcept
{
    {
        CHECK(hipFree(pGPUQWeight_));
        CHECK(hipFree(pGPUQBias_));
    }

    {
        CHECK(hipFree(pGPUKWeight_));
        CHECK(hipFree(pGPUKBias_));
    }

    {
        CHECK(hipFree(pGPUVWeight_));
        CHECK(hipFree(pGPUVBias_));
    }

    {
        CHECK(hipFree(pGPUOWeight_));
        CHECK(hipFree(pGPUOBias_));
    }

    {
        CHECK(hipFree(pGPUQKBias_));
        CHECK(hipFree(pGPUQCBias_));
    }

    WHERE_AM_I()
}

void CSMHAPlugin::destroy() noexcept
{
    WHERE_AM_I();
    if (bOwnWeight_)
    {
        {
            free(const_cast<void *>(qweight_.values));
            free(const_cast<void *>(qbias_.values));
        }

        {
            free(const_cast<void *>(kweight_.values));
            free(const_cast<void *>(kbias_.values));
        }

        {
            free(const_cast<void *>(vweight_.values));
            free(const_cast<void *>(vbias_.values));
        }

        {
            free(const_cast<void *>(oweight_.values));
            free(const_cast<void *>(obias_.values));
        }

        {
            free(const_cast<void *>(qkbias_.values));
            free(const_cast<void *>(qcbias_.values));
        }

    }
    CHECK(hipblasDestroy(handle_));
}

size_t CSMHAPlugin::getSerializationSize() const noexcept
{
    WHERE_AM_I()
    return sizeof(nK_) + sizeof(nN_)
        + sizeof(float) * qweight_.count + sizeof(float) * qbias_.count
        + sizeof(float) * kweight_.count + sizeof(float) * kbias_.count
        + sizeof(float) * vweight_.count + sizeof(float) * vbias_.count
        + sizeof(float) * oweight_.count + sizeof(float) * obias_.count
        + sizeof(float) * qkbias_.count + sizeof(float) * qcbias_.count;
}

void CSMHAPlugin::serialize(void *buffer) const noexcept
{
    WHERE_AM_I()
    char * data   = reinterpret_cast<char *>(buffer);
    size_t offset = 0;
    memcpy(data + offset, &nK_, sizeof(nK_));
    offset += sizeof(nK_);
    memcpy(data + offset, &nN_, sizeof(nN_));
    offset += sizeof(nN_);

    size_t size;

    {
        size = sizeof(float) * nK_ * nN_;
        memcpy(data + offset, qweight_.values, size);
        offset += size;
        size = sizeof(float) * nN_;
        memcpy(data + offset, qbias_.values, size);
        offset += size;
    }

    {
        size = sizeof(float) * nK_ * nN_;
        memcpy(data + offset, kweight_.values, size);
        offset += size;
        size = sizeof(float) * nN_;
        memcpy(data + offset, kbias_.values, size);
        offset += size;
    }

    {
        size = sizeof(float) * nK_ * nN_;
        memcpy(data + offset, vweight_.values, size);
        offset += size;
        size = sizeof(float) * nN_;
        memcpy(data + offset, vbias_.values, size);
        offset += size;
    }

    {
        size = sizeof(float) * nK_ * nN_;
        memcpy(data + offset, oweight_.values, size);
        offset += size;
        size = sizeof(float) * nN_;
        memcpy(data + offset, obias_.values, size);
        offset += size;
    }

    {
        size = sizeof(float) * nN_;
        memcpy(data + offset, qkbias_.values, size);
        offset += size;
        size = sizeof(float) * nN_;
        memcpy(data + offset, qcbias_.values, size);
        offset += size;
    }

}

void CSMHAPlugin::setPluginNamespace(const char *pluginNamespace) noexcept
{
    WHERE_AM_I()
    namespace_ = pluginNamespace;
}
const char *CSMHAPlugin::getPluginNamespace() const noexcept
{
    WHERE_AM_I()
    return namespace_.c_str();
}

const char *CSMHAPlugin::getPluginType() const noexcept
{
    WHERE_AM_I()
    return PLUGIN_NAME;
}

const char *CSMHAPlugin::getPluginVersion() const noexcept
{
    WHERE_AM_I()
    return PLUGIN_VERSION;
}

void CSMHAPlugin::attachToContext(cudnnContext *contextCudnn, cublasContext *contextCublas, IGpuAllocator *gpuAllocator) noexcept
{
    WHERE_AM_I()
    //handle_ = contextCublas;
}

void CSMHAPlugin::detachFromContext() noexcept
{
    WHERE_AM_I();
}

// class CSMHAPluginCreator

// Static class fields initialization
PluginFieldCollection    CSMHAPluginCreator::fc_ {};
std::vector<PluginField> CSMHAPluginCreator::attr_;

CSMHAPluginCreator::CSMHAPluginCreator()
{
    WHERE_AM_I()
    attr_.emplace_back(PluginField("k", nullptr, PluginFieldType::kINT32, 1));
    attr_.emplace_back(PluginField("n", nullptr, PluginFieldType::kINT32, 1));
    attr_.emplace_back(PluginField("qw", nullptr, PluginFieldType::kFLOAT32, 1));
    attr_.emplace_back(PluginField("qb", nullptr, PluginFieldType::kFLOAT32, 1));
    attr_.emplace_back(PluginField("kw", nullptr, PluginFieldType::kFLOAT32, 1));
    attr_.emplace_back(PluginField("kb", nullptr, PluginFieldType::kFLOAT32, 1));
    attr_.emplace_back(PluginField("vw", nullptr, PluginFieldType::kFLOAT32, 1));
    attr_.emplace_back(PluginField("vb", nullptr, PluginFieldType::kFLOAT32, 1));
    attr_.emplace_back(PluginField("ow", nullptr, PluginFieldType::kFLOAT32, 1));
    attr_.emplace_back(PluginField("ob", nullptr, PluginFieldType::kFLOAT32, 1));
    attr_.emplace_back(PluginField("qkb", nullptr, PluginFieldType::kFLOAT32, 1));
    attr_.emplace_back(PluginField("qcb", nullptr, PluginFieldType::kFLOAT32, 1));
    fc_.nbFields = attr_.size();
    fc_.fields   = attr_.data();
}

CSMHAPluginCreator::~CSMHAPluginCreator()
{
    WHERE_AM_I();
}

IPluginV2 *CSMHAPluginCreator::createPlugin(const char *name, const PluginFieldCollection *fc) noexcept
{
    WHERE_AM_I()

    printf("[CSMHA] [CSMHAPluginCreator] [createPlugin] %d\n",fc->nbFields);

    int     k, n;
    Weights qw, qb, kw, kb, vw, vb, ow, ob, qkb, qcb;
    for (int i = 0; i < fc->nbFields; i++)
    {
        PluginField field = fc->fields[i];
        std::string field_name(field.name);

        if (field_name.compare("qw") == 0)
        {
            qw.values = field.data;
            qw.count  = field.length;
            qw.type   = DataType::kFLOAT;
            continue;
        }
        if (field_name.compare("qb") == 0)
        {
            qb.values = field.data;
            qb.count  = field.length;
            qb.type   = DataType::kFLOAT;
            continue;
        }
        if (field_name.compare("kw") == 0)
        {
            kw.values = field.data;
            kw.count  = field.length;
            kw.type   = DataType::kFLOAT;
            continue;
        }
        if (field_name.compare("kb") == 0)
        {
            kb.values = field.data;
            kb.count  = field.length;
            kb.type   = DataType::kFLOAT;
            continue;
        }
        if (field_name.compare("vw") == 0)
        {
            vw.values = field.data;
            vw.count  = field.length;
            vw.type   = DataType::kFLOAT;
            continue;
        }
        if (field_name.compare("vb") == 0)
        {
            vb.values = field.data;
            vb.count  = field.length;
            vb.type   = DataType::kFLOAT;
            continue;
        }
        if (field_name.compare("ow") == 0)
        {
            ow.values = field.data;
            ow.count  = field.length;
            ow.type   = DataType::kFLOAT;
            continue;
        }
        if (field_name.compare("ob") == 0)
        {
            ob.values = field.data;
            ob.count  = field.length;
            ob.type   = DataType::kFLOAT;
            continue;
        }
        if (field_name.compare("qkb") == 0)
        {
            qkb.values = field.data;
            qkb.count  = field.length;
            qkb.type   = DataType::kFLOAT;
            continue;
        }
        if (field_name.compare("qcb") == 0)
        {
            qcb.values = field.data;
            qcb.count  = field.length;
            qcb.type   = DataType::kFLOAT;
            continue;
        }
        if (field_name.compare("k") == 0)
        {
            k = *reinterpret_cast<const int *>(field.data);
        }
        if (field_name.compare("n") == 0)
        {
            n = *reinterpret_cast<const int *>(field.data);
        }
    }
    return new CSMHAPlugin(name, qw, qb, kw, kb, vw, vb, ow, ob, qkb, qcb, k, n, true);
}

IPluginV2 *CSMHAPluginCreator::deserializePlugin(const char *name, const void *serialData, size_t serialLength) noexcept
{
    WHERE_AM_I()
    return new CSMHAPlugin(name, serialData, serialLength);
}

void CSMHAPluginCreator::setPluginNamespace(const char *pluginNamespace) noexcept
{
    WHERE_AM_I()
    namespace_ = pluginNamespace;
}

const char *CSMHAPluginCreator::getPluginNamespace() const noexcept
{
    WHERE_AM_I()
    return namespace_.c_str();
}

const char *CSMHAPluginCreator::getPluginName() const noexcept
{
    WHERE_AM_I()
    return PLUGIN_NAME;
}
const char *CSMHAPluginCreator::getPluginVersion() const noexcept
{
    WHERE_AM_I()
    return PLUGIN_VERSION;
}

const PluginFieldCollection *CSMHAPluginCreator::getFieldNames() noexcept
{
    WHERE_AM_I()
    return &fc_;
}

REGISTER_TENSORRT_PLUGIN(CSMHAPluginCreator);

} // namespace nvinfer1
